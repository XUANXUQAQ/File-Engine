#include "hip/hip_runtime.h"
﻿#include "pch.h"
#include <memory>
#include <string>
#include <vector>
#include <concurrent_unordered_map.h>
#include "cuda_copy_vector_util.h"
#include "kernels.cuh"
#include "cache.h"
#include "constans.h"
#include "hip/hip_runtime.h"
#include ""

inline void gpuAssert(hipError_t code, const char* file, int line)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		std::quick_exit(code);
	}
}

__device__ bool not_matched(char* path,
                            bool is_ignore_case,
                            char* keywords,
                            char* keywords_lower_case,
                            int keywords_length,
                            bool* is_keyword_path)
{
	for (int i = 0; i < keywords_length; ++i)
	{
		const bool is_keyword_path_val = is_keyword_path[i];
		char match_str[MAX_PATH_LENGTH];
		if (is_keyword_path_val)
		{
			get_parent_path(path, match_str);
		}
		else
		{
			get_file_name(path, match_str);
		}
		char* each_keyword;
		if (is_ignore_case)
		{
			each_keyword = keywords_lower_case + i * static_cast<unsigned long long>(MAX_PATH_LENGTH);
			// strlwr_cuda(each_keyword);
			strlwr_cuda(match_str);
		}
		else
		{
			each_keyword = keywords + i * static_cast<unsigned long long>(MAX_PATH_LENGTH);
		}
		if (strstr_cuda(match_str, each_keyword) == nullptr)
		{
			return true;
		}
	}
	return false;
}


__device__ int strcmp_cuda(const char* str1, const char* str2)
{
	while (*str1)
	{
		if (*str1 > *str2)return 1;
		if (*str1 < *str2)return -1;
		++str1;
		++str2;
	}
	if (*str1 < *str2)return -1;
	return 0;
}


__device__ char* strlwr_cuda(char* src)
{
	while (*src != '\0')
	{
		if (*src > 'A' && *src <= 'Z')
		{
			*src += 32;
		}
		++src;
	}
	return src;
}


__device__ char* strstr_cuda(char* s1, char* s2)
{
	int n;
	if (*s2) //两种情况考虑
	{
		while (*s1)
		{
			for (n = 0; *(s1 + n) == *(s2 + n); ++n)
			{
				if (!*(s2 + n + 1)) //查找的下一个字符是否为'\0'
				{
					return s1;
				}
			}
			++s1;
		}
		return nullptr;
	}
	return s1;
}

__device__ char* strrchr_cuda(const char* s, int c)
{
	if (s == nullptr)
	{
		return nullptr;
	}

	char* p_char = nullptr;
	while (*s != '\0')
	{
		if (*s == static_cast<char>(c))
		{
			p_char = const_cast<char*>(s);
		}
		++s;
	}

	return p_char;
}

__device__ char* strcpy_cuda(char* dst, const char* src)
{
	char* ret = dst;
	while ((*dst++ = *src++) != '\0')
	{
	}
	return ret;
}

__device__ void get_file_name(const char* path, char* output)
{
	const char* p = strrchr_cuda(path, '\\');
	strcpy_cuda(output, p + 1);
}

__device__ void get_parent_path(const char* path, char* output)
{
	strcpy_cuda(output, path);
	char* p = strrchr_cuda(output, '\\');
	*p = '\0';
}


__global__ void check(char* paths,
                      int* search_case,
                      bool* is_ignore_case,
                      char* search_text,
                      char* keywords,
                      char* keywords_lower_case,
                      size_t* keywords_length,
                      bool* is_keyword_path,
                      char* output)
{
	const int thread_id = GET_TID();
	output[thread_id] = 0;
	char* path = paths + thread_id * static_cast<unsigned long long>(MAX_PATH_LENGTH);
	if (path == nullptr || !path[0])
	{
		return;
	}
	if (not_matched(path, *is_ignore_case, keywords, keywords_lower_case, static_cast<int>(*keywords_length),
	                is_keyword_path))
	{
		return;
	}
	if (*search_case == 0)
	{
		output[thread_id] = 1;
		return;
	}
	// if ((*search_case & 1) == 1)
	// {
	// 	// TODO 判断是否为文件
	// }
	// if ((*search_case & 2) == 2)
	// {
	// 	// TODO 判断是否为文件夹
	// }
	if ((*search_case & 4) == 4)
	{
		// 全字匹配
		strlwr_cuda(search_text);
		char file_name[FILENAME_MAX];
		get_file_name(path, file_name);
		strlwr_cuda(file_name);
		if (strcmp_cuda(search_text, file_name) != 0)
		{
			return;
		}
	}
	output[thread_id] = 1;
}

void start_kernel(concurrency::concurrent_unordered_map<std::string, list_cache*>& cache_map,
                  const std::vector<std::string>& search_case,
                  bool is_ignore_case,
                  const char* search_text,
                  const std::vector<std::string>& keywords,
                  const std::vector<std::string>& keywords_lower_case,
                  const bool* is_keyword_path)
{
	int* dev_search_case = nullptr;
	char* dev_search_text = nullptr;
	char* dev_keywords = nullptr;
	char* dev_keywords_lower_case = nullptr;
	size_t* dev_keywords_length = nullptr;
	bool* dev_is_keyword_path = nullptr;
	bool* dev_is_ignore_case = nullptr;

	const auto keywords_num = keywords.size();
	const auto stream_count = cache_map.size();
	auto streams = new hipStream_t[stream_count];
	//初始化流
	for (size_t i = 0; i < stream_count; ++i)
	{
		gpuErrchk(hipStreamCreate(&streams[i]))
	}
	do
	{
		// 选择第一个GPU
		gpuErrchk(hipSetDevice(0))

		// 复制search case
		// 第一位为1表示有F，第二位为1表示有D，第三位为1表示有FULL，CASE由File-Engine主程序进行判断，传入参数is_ignore_case为false表示有CASE
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_search_case), sizeof(int)))
		int search_case_num = 0;
		for (auto& each_case : search_case)
		{
			//TODO 支持文件和文件夹判断
			// if (each_case == "f")
			// {
			// 	search_case_num |= 1;
			// }
			// if (each_case == "d")
			// {
			// 	search_case_num |= 2;
			// }
			if (each_case == "full")
			{
				search_case_num |= 4;
			}
		}
		gpuErrchk(hipMemcpy(dev_search_case, &search_case_num, sizeof(int), hipMemcpyHostToDevice))

		// 复制search text
		const auto search_text_len = strlen(search_text);
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_search_text), (search_text_len + 1) * sizeof(char)))
		gpuErrchk(hipMemset(dev_search_text, 0, search_text_len + 1))
		gpuErrchk(hipMemcpy(dev_search_text, search_text, search_text_len, hipMemcpyHostToDevice))

		// 复制keywords
		gpuErrchk(vector_to_cuda_char_array(keywords, reinterpret_cast<void**>(&dev_keywords), 0))

		// 复制keywords_lower_case
		gpuErrchk(vector_to_cuda_char_array(keywords_lower_case, reinterpret_cast<void**>(&dev_keywords_lower_case),0))

		//复制keywords_length
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_keywords_length), sizeof(size_t)))
		gpuErrchk(hipMemcpy(dev_keywords_length, &keywords_num, sizeof(size_t), hipMemcpyHostToDevice))

		// 复制is_keyword_path
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_is_keyword_path), sizeof(bool) * keywords_num))
		gpuErrchk(hipMemcpy(dev_is_keyword_path, is_keyword_path, sizeof(bool) * keywords_num, hipMemcpyHostToDevice))

		// 复制is_ignore_case
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_is_ignore_case), sizeof(bool)))
		gpuErrchk(hipMemcpy(dev_is_ignore_case, &is_ignore_case, sizeof(bool), hipMemcpyHostToDevice))
		int count = 0;
		for (auto& each : cache_map)
		{
			int block_num, thread_num;
			const auto& cache = each.second;
			if (cache->record_num > MAX_THREAD_PER_BLOCK)
			{
				thread_num = MAX_THREAD_PER_BLOCK;
				block_num = static_cast<int>(cache->record_num / thread_num);
			}
			else
			{
				thread_num = static_cast<int>(cache->record_num.load());
				block_num = 1;
			}
			cache->is_match_done = false;
			//注册回调
			hipStreamAddCallback(streams[count], set_match_done_flag_callback, cache, 0);

			check<<<block_num, thread_num, 0, streams[count]>>>
			(cache->dev_cache,
			 dev_search_case,
			 dev_is_ignore_case,
			 dev_search_text,
			 dev_keywords,
			 dev_keywords_lower_case,
			 dev_keywords_length,
			 dev_is_keyword_path,
			 cache->dev_output);
			++count;
		}

		// 检查启动错误
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "check launch failed: %s\n", hipGetErrorString(cudaStatus));
			break;
		}

		// 等待执行完成
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launch!\n", cudaStatus);
		}
	}
	while (false);
	delete[] streams;
	hipFree(dev_search_case);
	hipFree(dev_search_text);
	hipFree(dev_keywords);
	hipFree(dev_keywords_lower_case);
	hipFree(dev_is_keyword_path);
	hipFree(dev_is_ignore_case);
	hipFree(dev_keywords_length);
}

void CUDART_CB set_match_done_flag_callback(hipStream_t, hipError_t, void* data)
{
	const auto cache = static_cast<list_cache*>(data);
	cache->is_match_done = true;
}
