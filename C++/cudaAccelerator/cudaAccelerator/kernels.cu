#include "hip/hip_runtime.h"
﻿#include "pch.h"
#include <memory>
#include <string>
#include <vector>
#include <concurrent_unordered_map.h>
#include "cuda_copy_vector_util.h"
#include "kernels.cuh"
#include "cache.h"
#include "constans.h"
#include "hip/hip_runtime.h"
#include ""
#include "str_utils.cuh"
#include "str_convert.cuh"

inline void gpuAssert(hipError_t code, const char* file, int line, const char* function, bool is_exit,
                      const char* info)
{
	if (code != hipSuccess)
	{
		if (info == nullptr)
		{
			fprintf(stderr, "GPU assert: %s %s %d %s\n", hipGetErrorString(code), file, line, function);
		}
		else
		{
			fprintf(stderr, "GPU assert: %s %s %d %s %s\n", hipGetErrorString(code), file, line, function, info);
		}
		if (is_exit)
		{
			std::quick_exit(code);
		}
	}
}

__device__ void convert_to_pinyin(const char* chinese_str, char* output_str)
{
	static constexpr int spell_value[] = {
		-20319, -20317, -20304, -20295, -20292, -20283, -20265, -20257, -20242, -20230, -20051, -20036, -20032, -20026,
		-20002, -19990, -19986, -19982, -19976, -19805, -19784, -19775, -19774, -19763, -19756, -19751, -19746, -19741,
		-19739, -19728,
		-19725, -19715, -19540, -19531, -19525, -19515, -19500, -19484, -19479, -19467, -19289, -19288, -19281, -19275,
		-19270, -19263,
		-19261, -19249, -19243, -19242, -19238, -19235, -19227, -19224, -19218, -19212, -19038, -19023, -19018, -19006,
		-19003, -18996,
		-18977, -18961, -18952, -18783, -18774, -18773, -18763, -18756, -18741, -18735, -18731, -18722, -18710, -18697,
		-18696, -18526,
		-18518, -18501, -18490, -18478, -18463, -18448, -18447, -18446, -18239, -18237, -18231, -18220, -18211, -18201,
		-18184, -18183,
		-18181, -18012, -17997, -17988, -17970, -17964, -17961, -17950, -17947, -17931, -17928, -17922, -17759, -17752,
		-17733, -17730,
		-17721, -17703, -17701, -17697, -17692, -17683, -17676, -17496, -17487, -17482, -17468, -17454, -17433, -17427,
		-17417, -17202,
		-17185, -16983, -16970, -16942, -16915, -16733, -16708, -16706, -16689, -16664, -16657, -16647, -16474, -16470,
		-16465, -16459,
		-16452, -16448, -16433, -16429, -16427, -16423, -16419, -16412, -16407, -16403, -16401, -16393, -16220, -16216,
		-16212, -16205,
		-16202, -16187, -16180, -16171, -16169, -16158, -16155, -15959, -15958, -15944, -15933, -15920, -15915, -15903,
		-15889, -15878,
		-15707, -15701, -15681, -15667, -15661, -15659, -15652, -15640, -15631, -15625, -15454, -15448, -15436, -15435,
		-15419, -15416,
		-15408, -15394, -15385, -15377, -15375, -15369, -15363, -15362, -15183, -15180, -15165, -15158, -15153, -15150,
		-15149, -15144,
		-15143, -15141, -15140, -15139, -15128, -15121, -15119, -15117, -15110, -15109, -14941, -14937, -14933, -14930,
		-14929, -14928,
		-14926, -14922, -14921, -14914, -14908, -14902, -14894, -14889, -14882, -14873, -14871, -14857, -14678, -14674,
		-14670, -14668,
		-14663, -14654, -14645, -14630, -14594, -14429, -14407, -14399, -14384, -14379, -14368, -14355, -14353, -14345,
		-14170, -14159,
		-14151, -14149, -14145, -14140, -14137, -14135, -14125, -14123, -14122, -14112, -14109, -14099, -14097, -14094,
		-14092, -14090,
		-14087, -14083, -13917, -13914, -13910, -13907, -13906, -13905, -13896, -13894, -13878, -13870, -13859, -13847,
		-13831, -13658,
		-13611, -13601, -13406, -13404, -13400, -13398, -13395, -13391, -13387, -13383, -13367, -13359, -13356, -13343,
		-13340, -13329,
		-13326, -13318, -13147, -13138, -13120, -13107, -13096, -13095, -13091, -13076, -13068, -13063, -13060, -12888,
		-12875, -12871,
		-12860, -12858, -12852, -12849, -12838, -12831, -12829, -12812, -12802, -12607, -12597, -12594, -12585, -12556,
		-12359, -12346,
		-12320, -12300, -12120, -12099, -12089, -12074, -12067, -12058, -12039, -11867, -11861, -11847, -11831, -11798,
		-11781, -11604,
		-11589, -11536, -11358, -11340, -11339, -11324, -11303, -11097, -11077, -11067, -11055, -11052, -11045, -11041,
		-11038, -11024,
		-11020, -11019, -11018, -11014, -10838, -10832, -10815, -10800, -10790, -10780, -10764, -10587, -10544, -10533,
		-10519, -10331,
		-10329, -10328, -10322, -10315, -10309, -10307, -10296, -10281, -10274, -10270, -10262, -10260, -10256, -10254
	};

	// 395个字符串，每个字符串长度不超过6
	static constexpr char spell_dict[396][7] = {
		"a", "ai", "an", "ang", "ao", "ba", "bai", "ban", "bang", "bao", "bei", "ben", "beng", "bi", "bian", "biao",
		"bie", "bin", "bing", "bo", "bu", "ca", "cai", "can", "cang", "cao", "ce", "ceng", "cha", "chai", "chan",
		"chang", "chao", "che", "chen",
		"cheng", "chi", "chong", "chou", "chu", "chuai", "chuan", "chuang", "chui", "chun", "chuo", "ci", "cong", "cou",
		"cu", "cuan", "cui",
		"cun", "cuo", "da", "dai", "dan", "dang", "dao", "de", "deng", "di", "dian", "diao", "die", "ding", "diu",
		"dong", "dou", "du", "duan",
		"dui", "dun", "duo", "e", "en", "er", "fa", "fan", "fang", "fei", "fen", "feng", "fo", "fou", "fu", "ga", "gai",
		"gan", "gang", "gao",
		"ge", "gei", "gen", "geng", "gong", "gou", "gu", "gua", "guai", "guan", "guang", "gui", "gun", "guo", "ha",
		"hai", "han", "hang",
		"hao", "he", "hei", "hen", "heng", "hong", "hou", "hu", "hua", "huai", "huan", "huang", "hui", "hun", "huo",
		"ji", "jia", "jian",
		"jiang", "jiao", "jie", "jin", "jing", "jiong", "jiu", "ju", "juan", "jue", "jun", "ka", "kai", "kan", "kang",
		"kao", "ke", "ken",
		"keng", "kong", "kou", "ku", "kua", "kuai", "kuan", "kuang", "kui", "kun", "kuo", "la", "lai", "lan", "lang",
		"lao", "le", "lei",
		"leng", "li", "lia", "lian", "liang", "liao", "lie", "lin", "ling", "liu", "long", "lou", "lu", "lv", "luan",
		"lue", "lun", "luo",
		"ma", "mai", "man", "mang", "mao", "me", "mei", "men", "meng", "mi", "mian", "miao", "mie", "min", "ming",
		"miu", "mo", "mou", "mu",
		"na", "nai", "nan", "nang", "nao", "ne", "nei", "nen", "neng", "ni", "nian", "niang", "niao", "nie", "nin",
		"ning", "niu", "nong",
		"nu", "nv", "nuan", "nue", "nuo", "o", "ou", "pa", "pai", "pan", "pang", "pao", "pei", "pen", "peng", "pi",
		"pian", "piao", "pie",
		"pin", "ping", "po", "pu", "qi", "qia", "qian", "qiang", "qiao", "qie", "qin", "qing", "qiong", "qiu", "qu",
		"quan", "que", "qun",
		"ran", "rang", "rao", "re", "ren", "reng", "ri", "rong", "rou", "ru", "ruan", "rui", "run", "ruo", "sa", "sai",
		"san", "sang",
		"sao", "se", "sen", "seng", "sha", "shai", "shan", "shang", "shao", "she", "shen", "sheng", "shi", "shou",
		"shu", "shua",
		"shuai", "shuan", "shuang", "shui", "shun", "shuo", "si", "song", "sou", "su", "suan", "sui", "sun", "suo",
		"ta", "tai",
		"tan", "tang", "tao", "te", "teng", "ti", "tian", "tiao", "tie", "ting", "tong", "tou", "tu", "tuan", "tui",
		"tun", "tuo",
		"wa", "wai", "wan", "wang", "wei", "wen", "weng", "wo", "wu", "xi", "xia", "xian", "xiang", "xiao", "xie",
		"xin", "xing",
		"xiong", "xiu", "xu", "xuan", "xue", "xun", "ya", "yan", "yang", "yao", "ye", "yi", "yin", "ying", "yo", "yong",
		"you",
		"yu", "yuan", "yue", "yun", "za", "zai", "zan", "zang", "zao", "ze", "zei", "zen", "zeng", "zha", "zhai",
		"zhan", "zhang",
		"zhao", "zhe", "zhen", "zheng", "zhi", "zhong", "zhou", "zhu", "zhua", "zhuai", "zhuan", "zhuang", "zhui",
		"zhun", "zhuo",
		"zi", "zong", "zou", "zu", "zuan", "zui", "zun", "zuo"
	};
	// 循环处理字节数组
	const auto length = strlen_cuda(chinese_str);
	for (size_t j = 0; j < length;)
	{
		// 非汉字处理
		const unsigned char val = chinese_str[j];
		if (val < 128)
		{
			str_add_single(output_str, chinese_str[j]);
			// 偏移下标
			++j;
			continue;
		}

		// 汉字处理
		const int chrasc = chinese_str[j] * 256 + chinese_str[j + 1] + 256;
		if (chrasc > 0 && chrasc < 160)
		{
			// 非汉字
			str_add_single(output_str, chinese_str[j]);
			// 偏移下标
			++j;
		}
		else
		{
			// 汉字
			for (int i = sizeof spell_value / sizeof spell_value[0] - 1; i >= 0; --i)
			{
				// 查找字典
				if (spell_value[i] <= chrasc)
				{
					strcat_cuda(output_str, spell_dict[i]);
					break;
				}
			}
			// 偏移下标 （汉字双字节）
			j += 2;
		}
	}
}

__device__ bool not_matched(const char* path,
                            const bool is_ignore_case,
                            char* keywords,
                            char* keywords_lower_case,
                            const int keywords_length,
                            const bool* is_keyword_path)
{
	for (int i = 0; i < keywords_length; ++i)
	{
		const bool is_keyword_path_val = is_keyword_path[i];
		char match_str[MAX_PATH_LENGTH]{0};
		if (is_keyword_path_val)
		{
			get_parent_path(path, match_str);
		}
		else
		{
			get_file_name(path, match_str);
		}
		char* each_keyword;
		if (is_ignore_case)
		{
			each_keyword = keywords_lower_case + i * static_cast<unsigned long long>(MAX_PATH_LENGTH);
			strlwr_cuda(match_str);
		}
		else
		{
			each_keyword = keywords + i * static_cast<unsigned long long>(MAX_PATH_LENGTH);
		}
		if (!each_keyword[0])
		{
			continue;
		}
		if (!match_str[0] || strstr_cuda(match_str, each_keyword) == nullptr)
		{
			if (is_keyword_path_val || !is_str_contains_chinese(match_str))
			{
				return true;
			}
			char gbk_buffer[MAX_PATH_LENGTH * 2];
			char* gbk_buffer_ptr = gbk_buffer;
			unsigned gbk_buffer_size = MAX_PATH_LENGTH * 2;
			// utf-8编码转换gbk
			str_normalize_init();
			utf8_to_gbk(match_str, static_cast<unsigned>(strlen_cuda(match_str)), &gbk_buffer_ptr, &gbk_buffer_size);
			char converted_pinyin[MAX_PATH_LENGTH * 6]{0};
			convert_to_pinyin(gbk_buffer, converted_pinyin);
			if (strstr_cuda(converted_pinyin, each_keyword) == nullptr)
			{
				return true;
			}
		}
	}
	return false;
}

__global__ void check(const char (* str_address_ptr_array)[MAX_PATH_LENGTH],
                      const int* search_case,
                      const bool* is_ignore_case,
                      char* search_text,
                      char* keywords,
                      char* keywords_lower_case,
                      const size_t* keywords_length,
                      const bool* is_keyword_path,
                      char* output,
                      const bool* is_stop_collect_var)
{
	const size_t thread_id = GET_TID();
	const char* path = reinterpret_cast<const char*>(str_address_ptr_array + thread_id);
	if (*is_stop_collect_var)
	{
		return;
	}
#ifdef DEBUG_OUTPUT
	printf("%s\n", path);
#endif
	if (path == nullptr || !path[0])
	{
		return;
	}
	if (not_matched(path, *is_ignore_case, keywords, keywords_lower_case, static_cast<int>(*keywords_length),
	                is_keyword_path))
	{
		return;
	}
	if (*search_case == 0)
	{
		output[thread_id] = 1;
		return;
	}
	if ((*search_case & 4) == 4)
	{
		// 全字匹配
		strlwr_cuda(search_text);
		char file_name[MAX_PATH_LENGTH];
		get_file_name(path, file_name);
		strlwr_cuda(file_name);
		if (strcmp_cuda(search_text, file_name) != 0)
		{
			return;
		}
	}
	output[thread_id] = 1;
}

void start_kernel(concurrency::concurrent_unordered_map<std::string, list_cache*>& cache_map,
                  const std::vector<std::string>& search_case,
                  bool is_ignore_case,
                  const char* search_text,
                  const std::vector<std::string>& keywords,
                  const std::vector<std::string>& keywords_lower_case,
                  const bool* is_keyword_path,
                  hipStream_t* streams,
                  const size_t stream_count)
{
	int* dev_search_case = nullptr;
	char* dev_search_text = nullptr;
	char* dev_keywords = nullptr;
	char* dev_keywords_lower_case = nullptr;
	size_t* dev_keywords_length = nullptr;
	bool* dev_is_keyword_path = nullptr;
	bool* dev_is_ignore_case = nullptr;

	const auto keywords_num = keywords.size();

	//初始化流
	for (size_t i = 0; i < stream_count; ++i)
	{
		gpuErrchk(hipStreamCreate(&streams[i]), true, nullptr)
	}
	do
	{
		// 选择第一个GPU
		gpuErrchk(hipSetDevice(0), true, nullptr)

		// 复制search case
		// 第一位为1表示有F，第二位为1表示有D，第三位为1表示有FULL，CASE由File-Engine主程序进行判断，传入参数is_ignore_case为false表示有CASE
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_search_case), sizeof(int)), true, nullptr)
		int search_case_num = 0;
		for (auto& each_case : search_case)
		{
			// if (each_case == "f")
			// {
			// 	search_case_num |= 1;
			// }
			// if (each_case == "d")
			// {
			// 	search_case_num |= 2;
			// }
			if (each_case == "full")
			{
				search_case_num |= 4;
			}
		}
		gpuErrchk(hipMemcpy(dev_search_case, &search_case_num, sizeof(int), hipMemcpyHostToDevice), true, nullptr)

		// 复制search text
		const auto search_text_len = strlen(search_text);
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_search_text), (search_text_len + 1) * sizeof(char)), true,
		          nullptr)
		gpuErrchk(hipMemset(dev_search_text, 0, search_text_len + 1), true, nullptr)
		gpuErrchk(hipMemcpy(dev_search_text, search_text, search_text_len, hipMemcpyHostToDevice), true, nullptr)

		// 复制keywords
		gpuErrchk(vector_to_cuda_char_array(keywords, reinterpret_cast<void**>(&dev_keywords)), true, nullptr)

		// 复制keywords_lower_case
		gpuErrchk(vector_to_cuda_char_array(keywords_lower_case, reinterpret_cast<void**>(&dev_keywords_lower_case)),
		          true, nullptr)

		//复制keywords_length
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_keywords_length), sizeof(size_t)), true, nullptr)
		gpuErrchk(hipMemcpy(dev_keywords_length, &keywords_num, sizeof(size_t), hipMemcpyHostToDevice), true, nullptr)

		// 复制is_keyword_path
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_is_keyword_path), sizeof(bool) * keywords_num), true,
		          nullptr)
		gpuErrchk(hipMemcpy(dev_is_keyword_path, is_keyword_path, sizeof(bool) * keywords_num, hipMemcpyHostToDevice),
		          true, nullptr)

		// 复制is_ignore_case
		gpuErrchk(hipMalloc(reinterpret_cast<void**>(&dev_is_ignore_case), sizeof(bool)), true, nullptr)
		gpuErrchk(hipMemcpy(dev_is_ignore_case, &is_ignore_case, sizeof(bool), hipMemcpyHostToDevice), true, nullptr)
		int count = 0;
		for (auto& each : cache_map)
		{
			const auto& cache = each.second;
			if (!cache->is_cache_valid)
			{
				continue;
			}
			int block_num, thread_num;
			if (cache->str_data.record_num > MAX_THREAD_PER_BLOCK)
			{
				thread_num = MAX_THREAD_PER_BLOCK;
				block_num = static_cast<int>(cache->str_data.record_num / thread_num);
			}
			else
			{
				thread_num = static_cast<int>(cache->str_data.record_num.load());
				block_num = 1;
			}
			//注册回调
			hipStreamAddCallback(streams[count], set_match_done_flag_callback, cache, 0);

			check<<<block_num, thread_num, 0, streams[count]>>>
			(cache->str_data.dev_cache_str,
			 dev_search_case,
			 dev_is_ignore_case,
			 dev_search_text,
			 dev_keywords,
			 dev_keywords_lower_case,
			 dev_keywords_length,
			 dev_is_keyword_path,
			 cache->dev_output,
			 get_dev_stop_signal());
			++count;
		}

		// 检查启动错误
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "check launch failed: %s\n", hipGetErrorString(cudaStatus));
			break;
		}
	}
	while (false);
	hipFree(dev_search_case);
	hipFree(dev_search_text);
	hipFree(dev_keywords);
	hipFree(dev_keywords_lower_case);
	hipFree(dev_is_keyword_path);
	hipFree(dev_is_ignore_case);
	hipFree(dev_keywords_length);
}

void CUDART_CB set_match_done_flag_callback(hipStream_t, hipError_t, void* data)
{
	const auto cache = static_cast<list_cache*>(data);
	cache->is_match_done = true;
}
